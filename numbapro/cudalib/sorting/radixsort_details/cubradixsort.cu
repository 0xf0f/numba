#include <cub/device/device_radix_sort.cuh>
#include <stdint.h>

struct TempStorage{
    void * storage;
    size_t storage_bytes;
};

static
void cleanup(TempStorage *ptr) {
    hipFree(ptr->storage);
    delete ptr;
}

template <class Tk, class Tv=unsigned>
struct RadixSort {


    static
    TempStorage* sort(  TempStorage *temp,
                        unsigned  num_items,
                        Tk  *d_key_buf,
                        Tk  *d_key_alt_buf,
                        Tv  *d_value_buf,
                        Tv  *d_value_alt_buf,
                        hipStream_t stream,
                        int descending,
                        unsigned begin_bit,
                        unsigned end_bit      )
    {
        hipcub::DoubleBuffer<Tk> d_keys(d_key_buf, d_key_alt_buf);
        if (temp == 0) {
            temp = new TempStorage;
            temp->storage = 0;
            temp->storage_bytes = 0;
        }
        if (d_value_buf) {
            // Sort KeyValue pairs
            hipcub::DoubleBuffer<Tv> d_values(d_value_buf, d_value_alt_buf);
            if (descending) {
                hipcub::DeviceRadixSort::SortPairsDescending(temp->storage,
                                                          temp->storage_bytes,
                                                          d_keys,
                                                          d_values,
                                                          num_items,
                                                          begin_bit,
                                                          end_bit,
                                                          stream);
            } else {
                hipcub::DeviceRadixSort::SortPairs(  temp->storage,
                                                  temp->storage_bytes,
                                                  d_keys,
                                                  d_values,
                                                  num_items,
                                                  begin_bit,
                                                  end_bit,
                                                  stream    );
            }

            if (temp->storage && d_value_buf != d_values.Current()){
                hipMemcpyAsync(d_value_buf, d_value_alt_buf,
                                num_items * sizeof(Tv),
                                hipMemcpyDeviceToDevice,
                                stream);
            }
        } else {
            // Sort Keys only
            if (descending) {
                hipcub::DeviceRadixSort::SortKeysDescending(   temp->storage,
                                                            temp->storage_bytes,
                                                            d_keys,
                                                            num_items,
                                                            begin_bit,
                                                            end_bit,
                                                            stream  );
            } else {
                hipcub::DeviceRadixSort::SortKeys( temp->storage,
                                                temp->storage_bytes,
                                                d_keys,
                                                num_items,
                                                begin_bit,
                                                end_bit,
                                                stream  );
            }
        }

        if (temp->storage && d_key_buf != d_keys.Current()){
            hipMemcpyAsync(d_key_buf, d_key_alt_buf, num_items * sizeof(Tk),
                            hipMemcpyDeviceToDevice, stream);
        }

        if (temp->storage == 0) {
            hipMalloc(&temp->storage, temp->storage_bytes);
            return temp;
        }
        return temp;
    }
};

extern "C" {

#define WRAP(Fn, Tk, Tv)                        \
void                                            \
radixsort_ ## Fn(   TempStorage *temp,          \
                    unsigned  num_items,        \
                    Tk  *d_key_buf,             \
                    Tk  *d_key_alt_buf,         \
                    Tv  *d_value_buf,           \
                    Tv  *d_value_alt_buf,       \
                    hipStream_t stream,        \
                    int descending,             \
                    unsigned begin_bit,         \
                    unsigned end_bit      ) {   \
    RadixSort<Tk, Tv>::sort(temp,               \
                            num_items,          \
                            d_key_buf,          \
                            d_key_alt_buf,      \
                            d_value_buf,        \
                            d_value_alt_buf,    \
                            stream,             \
                            descending,         \
                            begin_bit,          \
                            end_bit);           \
}

WRAP(float, float, unsigned)
WRAP(double, double, unsigned)
WRAP(int32, int32_t, unsigned)
WRAP(uint32, uint32_t, unsigned)
WRAP(int64, int64_t, unsigned)
WRAP(uint64, uint64_t, unsigned)

void
radixsort_cleanup(TempStorage *ptr) {
    cleanup(ptr);
}

#undef WRAP
} // end extern "C"
