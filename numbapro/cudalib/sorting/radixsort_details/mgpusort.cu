#include <moderngpu.cuh>
#include <util/mgpucontext.h>
#include <src/mgpucontext.cu>
// #include <src/mgpuutil.cpp>

namespace mgpu{
	std::string stringprintf(const char* format, ...) { return std::string(); }
}

namespace {

using namespace mgpu;

template<class Tkey, class Tval>
void segsortpairs( Tkey *d_keys,
				   Tval *d_vals,
				   unsigned N,
				   const int *d_segments,
				   unsigned NumSegs,
				   hipStream_t stream	)
{

    ContextPtr context = CreateCudaDeviceAttachStream(stream);

    SegSortPairsFromIndices(
    	d_keys,
    	d_vals,
    	N,
    	d_segments,
    	NumSegs,
    	*context,
    	false);

}

} // end static namespace


extern "C" {

#define WRAP(F, Tkey, Tval)												\
void segsortpairs_##F( Tkey *d_keys,										\
					   Tval *d_vals,										\
					   unsigned N,										\
					   const int *d_segments,								\
					   unsigned NumSegs,								\
					   hipStream_t stream	)							\
{  segsortpairs(d_keys, d_vals, N, d_segments, NumSegs, stream);  }

WRAP(float32, float, unsigned)
WRAP(float64, float, unsigned)


}
