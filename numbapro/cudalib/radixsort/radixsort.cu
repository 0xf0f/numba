/**
Divide into 256 element block.

For each block:
- build histogram

**/
// compile with: nvcc -arch=sm_30 radixsort_better.cu -I./cub

#include <iostream>
#include "radixutils.h"

int
main()
{
	using std::cout;
	using std::endl;

    typedef uint32_t data_type;

	const unsigned stride = sizeof(data_type);

	unsigned ct_data = 258;
	unsigned sz_data = sizeof(data_type) * ct_data;

    const unsigned ct_block = (ct_data + (BUCKET_SIZE-1)) / BUCKET_SIZE;
    cout << "ct_block = " << ct_block << '\n';

	unsigned ct_hist = ct_block * BUCKET_SIZE;
	unsigned sz_hist = sizeof(unsigned) * ct_hist;

    unsigned ct_bucket_total = BUCKET_SIZE;
    unsigned sz_bucket_total = sizeof(unsigned) * ct_bucket_total;

    unsigned sz_indices = sizeof(unsigned)*ct_data;

	data_type *data = new data_type[ct_data];
	unsigned *hist = new unsigned[ct_hist];
    unsigned *bucket_total = new unsigned[ct_bucket_total];
    unsigned *indices = new unsigned[ct_data];

	for (unsigned i=0; i<ct_data; ++i) {
		data[i] = i;// ct_data - i - 1;
	}

    uint8_t *dev_data;
    uint8_t *dev_sorted;
    unsigned *dev_hist;
    unsigned *dev_bucket_total;
    unsigned *dev_indices;

    hipMalloc(&dev_data, sz_data);
    hipMalloc(&dev_indices, sz_data);
    hipMalloc(&dev_sorted, sz_indices);
    hipMalloc(&dev_hist, sz_hist);
    hipMalloc(&dev_bucket_total, sz_bucket_total);

    // send data

	hipMemcpy(dev_data, data, sz_data, hipMemcpyHostToDevice);
	ASSERT_CUDA_LAST_ERROR();


    // compute

    for (unsigned offset=0; offset < 4; ++offset) {

    	cu_build_histogram<<<ct_block, BUCKET_SIZE>>>(
    		dev_data,
    		dev_hist,
    		stride,
    		offset,
    		ct_data
    	);
    	ASSERT_CUDA_LAST_ERROR();

        cu_scan_histogram<<<BUCKET_SIZE, SCAN_HISTOGRAM_BLOCK_SIZE>>>(
            dev_hist,
            dev_bucket_total,
            ct_block
        );
        ASSERT_CUDA_LAST_ERROR();


        cu_scan_bucket_index<<<1, BUCKET_SIZE>>>(dev_bucket_total);
        ASSERT_CUDA_LAST_ERROR();

        compute_indices(
            dev_data,
            dev_indices,
            dev_hist,
            dev_bucket_total,
            ct_data,
            stride,
            offset,
            ct_block
        );

        hipMemcpy(indices, dev_indices, sz_data, hipMemcpyDeviceToHost);
    	ASSERT_CUDA_LAST_ERROR();
    	for (int i=0; i<ct_data; ++i) {
            cout << " " << indices[i];
    	}
    	cout << endl;



        cu_scatter<<<ct_block, BUCKET_SIZE>>>(
            dev_data,
            dev_sorted,
            dev_indices,
            ct_data,
            stride
        );

        // cu_scatter_histogram_naive<<<ct_block, 1>>>(
        //     dev_data,
        //     dev_sorted,
        //     dev_hist,
        //     dev_bucket_total,
        //     ct_data,
        //     stride,
        //     offset
        // );

        // cu_scatter_histogram<<<ct_block, BUCKET_SIZE>>>(
        //     dev_data,
        //     dev_sorted,
        //     dev_hist,
        //     dev_bucket_total,
        //     ct_data,
        //     stride,
        //     offset
        // );
        ASSERT_CUDA_LAST_ERROR();

        hipMemcpy(dev_data, dev_sorted, sz_data, hipMemcpyDeviceToDevice);
        ASSERT_CUDA_LAST_ERROR();
    }
    // write back

 //    hipMemcpy(hist, dev_hist, sz_hist, hipMemcpyDeviceToHost);
 //    ASSERT_CUDA_LAST_ERROR();


 //    hipMemcpy(bucket_total, dev_bucket_total, sz_bucket_total,
 //               hipMemcpyDeviceToHost);
	// ASSERT_CUDA_LAST_ERROR();

    hipMemcpy(data, dev_sorted, sz_data, hipMemcpyDeviceToHost);
    ASSERT_CUDA_LAST_ERROR();

    // cout << "hist\n";
    // for (unsigned i=0; i<BUCKET_SIZE; ++i) {
    //     for (unsigned j=0; j<ct_block; ++j) {
    //        cout << "bucket " << i << " block " << j
    //             << " = " << hist[ct_block * i + j] << '\n';
    //     }
    // }
    // cout << "bucket total\n";
    // for (unsigned i=0; i<ct_bucket_total; ++i) {
    //     cout << i << ' ' << bucket_total[i] << '\n';
    // }

    cout << "sorted\n";
    for (unsigned i = 0; i < ct_data; ++i) {
        // cout << i << ' ' << data[i] << '\n';
        if(data[i] != i) {
            cout << "error at i = " << i << " = " << data[i] << endl;
            exit(1);
        }
    }


	cout << "ok" << endl;
	return 0;
}
