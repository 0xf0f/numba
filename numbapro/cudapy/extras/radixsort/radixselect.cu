/**
Divide into 256 element block.

For each block:
- build histogram

**/
// compile with: nvcc -arch=sm_30 radixsort_better.cu -I./cub

#include <iostream>
#include "radixutils.h"

void
radix_select(
    uint8_t  *dev_data,
    uint8_t  *dev_sorted,
    unsigned *dev_hist,
    unsigned *dev_bucket_total,
    unsigned *dev_indices,
    unsigned ct_block,
    unsigned ct_data,
    unsigned sz_data,
    unsigned sz_bucket_total,
    unsigned ct_bucket_total,
    unsigned stride,
    unsigned select_count,
    unsigned offset)
{
    using namespace std;
    unsigned bucket_total[ct_bucket_total];

    cu_build_histogram<<<ct_block, BUCKET_SIZE>>>(
        dev_data,
        dev_hist,
        stride,
        offset,
        ct_data
    );
    ASSERT_CUDA_LAST_ERROR();

    cu_scan_histogram<<<BUCKET_SIZE, SCAN_HISTOGRAM_BLOCK_SIZE>>>(
        dev_hist,
        dev_bucket_total,
        ct_block
    );
    ASSERT_CUDA_LAST_ERROR();

    hipMemcpy(bucket_total, dev_bucket_total, sz_bucket_total,
               hipMemcpyDeviceToHost);
    ASSERT_CUDA_LAST_ERROR();

    cu_scan_bucket_index<<<1, BUCKET_SIZE>>>(dev_bucket_total);
    ASSERT_CUDA_LAST_ERROR();


    compute_indices(
        dev_data,
        dev_indices,
        dev_hist,
        dev_bucket_total,
        ct_data,
        stride,
        offset,
        ct_block
    );

    cu_scatter<<<ct_block, BUCKET_SIZE>>>(
        dev_data,
        dev_sorted,
        dev_indices,
        ct_data,
        stride
    );

    ASSERT_CUDA_LAST_ERROR();

    hipMemcpy(dev_data, dev_sorted, sz_data, hipMemcpyDeviceToDevice);
    ASSERT_CUDA_LAST_ERROR();

    unsigned total_data = 0;

    if (offset == 0) return;

    for(unsigned i=0; i<ct_bucket_total && total_data < select_count; ++i) {

        unsigned sub_ct_data = bucket_total[i];
        unsigned sub_ct_block = (sub_ct_data + (BUCKET_SIZE-1)) / BUCKET_SIZE;
        unsigned sub_sz_data = sub_ct_data * stride;
        unsigned sub_data_offset = total_data * stride;

        radix_select(sub_data_offset + dev_data,
                     sub_data_offset + dev_sorted,
                     dev_hist,
                     dev_bucket_total,
                     dev_indices,
                     sub_ct_block,
                     sub_ct_data,
                     sub_sz_data,
                     sz_bucket_total,
                     ct_bucket_total,
                     stride,
                     select_count,
                     offset - 1);

        total_data += sub_ct_data;
    }
}


int
main()
{
	using std::cout;
	using std::endl;

    const int select_count = 100;

    typedef uint32_t data_type;

	const unsigned stride = sizeof(data_type);

	unsigned ct_data = 1000;
	unsigned sz_data = sizeof(data_type) * ct_data;

    unsigned ct_block = (ct_data + (BUCKET_SIZE-1)) / BUCKET_SIZE;
    cout << "ct_block = " << ct_block << '\n';

	unsigned ct_hist = ct_block * BUCKET_SIZE;
	unsigned sz_hist = sizeof(unsigned) * ct_hist;

    unsigned ct_bucket_total = BUCKET_SIZE;
    unsigned sz_bucket_total = sizeof(unsigned) * ct_bucket_total;

    unsigned sz_indices = sizeof(unsigned)*ct_data;

	data_type *data = new data_type[ct_data];
	unsigned *hist = new unsigned[ct_hist];

	for (unsigned i=0; i<ct_data; ++i) {
		data[i] = ct_data - i - 1;
	}

    uint8_t *dev_data;
    uint8_t *dev_sorted;
    unsigned *dev_hist;
    unsigned *dev_bucket_total;
    unsigned *dev_indices;

    hipMalloc(&dev_data, sz_data);
    hipMalloc(&dev_indices, sz_data);
    hipMalloc(&dev_sorted, sz_indices);
    hipMalloc(&dev_hist, sz_hist);
    hipMalloc(&dev_bucket_total, sz_bucket_total);

    // send data

	hipMemcpy(dev_data, data, sz_data, hipMemcpyHostToDevice);
	ASSERT_CUDA_LAST_ERROR();

    // compute

    radix_select(dev_data,
                 dev_sorted,
                 dev_hist,
                 dev_bucket_total,
                 dev_indices,
                 ct_block,
                 ct_data,
                 sz_data,
                 sz_bucket_total,
                 ct_bucket_total,
                 stride,
                 select_count,
                 3);

    // write back

    hipMemcpy(data, dev_sorted, sz_data, hipMemcpyDeviceToHost);
    ASSERT_CUDA_LAST_ERROR();

    cout << "sorted\n";
    for (unsigned i = 0; i < select_count; ++i) {
        cout << i << ' ' << data[i] << '\n';
        if(data[i] != i) {
            cout << "error at i = " << i << " = " << data[i] << endl;
            exit(1);
        }
    }


	cout << "ok" << endl;
	return 0;
}